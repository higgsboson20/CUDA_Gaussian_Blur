
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define PI 3.14159265358979323846

#define DIV_ROUND_UP(n, d)  (((n) + (d) - 1) / (d))

#define cuda_check(ret) _cuda_check((ret), __FILE__, __LINE__)

inline void _cuda_check(hipError_t ret, const char *file, int line)
{
        if (ret != hipSuccess) {
                fprintf(stderr, "CudaErr: %s (%s:%d)\n", hipGetErrorString(ret), file, line);
                exit(1);
        }
}

__global__ void convolution_kernel(int kernel_order, float * kernel_matrix, unsigned char * image_in, unsigned char * image_out, size_t width, size_t height)
{
    size_t row = blockIdx.y*blockDim.y + threadIdx.y;
    size_t col = blockIdx.x*blockDim.x + threadIdx.x;

    int dc = (kernel_order - 1)/2;

    float conv = 0, sum = 0;

    for(size_t k_i = 0; k_i < kernel_order; k_i++){
        for(size_t k_j = 0; k_j < kernel_order; k_j++){
            int ii = row + k_i - dc;
            int jj = col + k_j - dc;

            // Handle boundaries by clamping to the nearest edge pixel
            if (ii < 0) ii = 0;
            if (ii >= height) ii = height - 1;
            if (jj < 0) jj = 0;
            if (jj >= width) jj = width - 1;

            conv += image_in[width*ii + jj]*kernel_matrix[kernel_order*k_i + k_j];
            sum += kernel_matrix[kernel_order*k_i + k_j];

        }
    }

    if(sum > 0)
        conv /= sum;

    image_out[row*width + col] = (unsigned char)conv;

}

int main(int argc, char * argv[])
{
    /**To ensure there are three arguments: <input_pgm>, <output_pgm> and <sigma>*/
    if (argc != 4) {
         fprintf(stderr, "Usage: ./gaussian_blur_serial <input_pgm> <output_pgm> <sigma>\n");
    }


    // =================================================
    // Fill initial image buffer, initialize out image buffer, and create kernel matrix
    // =================================================

    /**Parse the command line to recieve the input file, output file name and the sigma value*/
    const char* input_file = argv[1]; //input binary PGM file
    const char* output_file = argv[2]; //output binary PGM file
    float sigma = atof(argv[3]); //sigma value

    //char temp[200]; // temporary string to hold a copy of the input filename
    size_t width, height;
    unsigned char max_value;

    /**Open the input binary PGM file*/
    FILE* file = fopen(input_file, "rb");
    if (file == NULL) {
        perror("Cannot open file");
        exit(1);
    }

    if (fscanf(file, "P5\n%ld %ld\n%hhd\n", &width, &height, &max_value) != 3) {
        fprintf(stderr, "Error reading image header\n");
        fclose(file);
        return EXIT_FAILURE;
    }

    unsigned char * image = (unsigned char *)malloc(width*height); // image binary data
    unsigned char * blurred_image = (unsigned char *)malloc(width*height); // blurred image
    /** Push image into buffer */
    uint32_t seek_dist = 3 + (int)log10(height) + (int)log10(width) + 4 + 4; // |firstLine| + |secondLine| + |thirdLine|
    fseek(file, seek_dist, SEEK_SET);

    if (fread(image, 1, height * width, file) != height * width) {
        fprintf(stderr, "Error reading image data\n");
        free(image);
        fclose(file);
        return 1;
    }

    /**Validate the bounds of sigma*/
    if (sigma <= 0) {
        fprintf(stderr, "Sigma value must be greater than 0\n");
        exit(1);
    }

    /**Create the gaussian kernel matrix*/
    uint32_t kernel_order = ceil(sigma * 6); //order of the kernel matrix
    uint32_t half_order = kernel_order / 2;

    if (kernel_order % 2 == 0) kernel_order += 1; //make it odd to account for edges

    // float** kernel_matrix = create_kernel_matrix(kernel_order, sigma);
    float * kernel_matrix  = (float *)malloc(sizeof(float) * kernel_order * kernel_order);
    float normalizer = 1 / (2 * PI * sigma * sigma);
    float sum = 0.0;

    //fills in the gaussian kernel matrix
    for (size_t x = 0; x < kernel_order; x++) {
        for (size_t y = 0; y < kernel_order; y++) {
            uint32_t x_offset = x - half_order;
            uint32_t y_offset = y - half_order;
            kernel_matrix[kernel_order*x + y] = normalizer * exp(-1.0 * (x_offset * x_offset + y_offset * y_offset) / (2.0 * sigma * sigma));
            sum += kernel_matrix[kernel_order*x + y];
        }
    }

    /** normalizes the gaussian kernel matrix to the total sum */
    for (size_t i = 0; i < kernel_order; i++) {
        for (size_t j = 0; j < kernel_order; j++) {
            kernel_matrix[kernel_order*i + j] /= sum;
        }
    }

    // =================================================
    //
    // =================================================


    // =================================================
    // initialize CUDA business
    // =================================================

    unsigned char * image_in, * image_out;
    float * kernel_matrix_d;

    cuda_check(hipMalloc(&kernel_matrix_d, kernel_order*kernel_order*sizeof(float)));
    cuda_check(hipMalloc(&image_in, width*height));
    cuda_check(hipMalloc(&image_out, width*height));

    cuda_check(hipMemcpy(kernel_matrix_d, kernel_matrix, kernel_order*kernel_order*sizeof(float), hipMemcpyHostToDevice));
    cuda_check(hipMemcpy(image_in, image, width*height, hipMemcpyHostToDevice));

    dim3 block_dim(32,32);
    dim3 grid_dim(DIV_ROUND_UP(width, block_dim.x), DIV_ROUND_UP(height, block_dim.y));
    convolution_kernel<<<grid_dim, block_dim>>>(kernel_order, kernel_matrix_d, image_in, image_out, width, height);

    cuda_check(hipPeekAtLastError());
    cuda_check(hipDeviceSynchronize());

    cuda_check(hipMemcpy(blurred_image, image_out, width*height, hipMemcpyDeviceToHost));

    cuda_check(hipFree(image_in));
    cuda_check(hipFree(image_out));


    // =================================================
    //
    // =================================================

    /** output file */
    FILE * out = fopen(output_file, "wb");
    if(!out){
        fprintf(stderr, "Error with opening file!\n");
        exit(1);
    }
    fprintf(out, "P5\n%ld %ld\n%d\n", width, height, max_value);

    fwrite(blurred_image, sizeof(char), width*height, out);

    fclose(out);
    fclose(file);

    free(image);
    free(kernel_matrix);
    free(blurred_image);



    return 0;

}
